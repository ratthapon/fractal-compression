
#include <hip/hip_runtime.h>

extern "C"
__global__ void limitCoeff
(int nBatch,int rbs, float maxCoeff,
	float *DA, float *RA, float *CA)
	{
		int taskIdx = blockIdx.x * blockDim.x + threadIdx.x;
		if (taskIdx < nBatch)
		{
			int i = taskIdx; // % (nBatch / 2);

			// support only 2 coefficients
			int nCoeff = 2;

			// locate arrays pointer
			int daOffset = i * rbs * nCoeff;
			int raOffset = i * rbs;
			int caOffset = i * nCoeff; // support only 2 coefficients

			// check if need to refit coefficients
			if (CA[caOffset + 1] > maxCoeff || CA[caOffset + 1] < -maxCoeff) {
				// set to maximum or minimum depend on sign
				if (CA[caOffset + 1] > maxCoeff) {
					CA[caOffset + 1] = maxCoeff;
				} else if (CA[caOffset + 1] < -maxCoeff) {
					CA[caOffset + 1] = -maxCoeff;
				}

				// refit coefficients
				float suma = 0.0f; // power 1 coeff
				float sumb = 0.0f; // power 0 coeff
				for(int j = 0; j<rbs ;j++){
					suma += DA[daOffset + rbs + j];
					sumb += RA[raOffset + j];
				}
				CA[caOffset] = (sumb - CA[caOffset + 1] * suma) / rbs;
			}
		}
	}

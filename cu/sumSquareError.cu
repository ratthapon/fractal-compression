
#include <hip/hip_runtime.h>

extern "C"
__global__ void sumSquareError
(int nBatch,int rbs, int nCoeff,
float *DA, float *CA, float *EA, float *SA)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nBatch)
    {
		SA[i] = 0;
		for(int j = 0; j < rbs ; j++){
			float fx = 0.0f;
			for(int k = 0 ; k < nCoeff ; k++){
				fx += DA[i*rbs*nCoeff + rbs*k + j] * CA[i*nCoeff + k];
			}
			float error = EA[i*rbs + j] - fx;
			//EA[i*rbs + j] = error; // store error value
			SA[i] += error*error; // sum square error
		}
    }
}


#include <hip/hip_runtime.h>
extern "C"
__global__ void setDomainPoolKernel(
int nBatch,int rbs,int nDegree,int nDScale,int dBaseScale, float regularize,

float *data,float *dataRev, // array of data and reverse data
// arrays pointer
float *DA,
float *AA,
float *IA,
// pointer of array of pointer to pointer of array in arrays, nevermind i just stun you.
// p(i) = data(i + size(data))
float **DP,
float **AP,
float **IP
)
{
    int taskIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (taskIdx < nBatch)
    {
		// initialize domain arrays
		int nCoeff = ((nDegree - 1) * nDScale + 1);
		int dpOffset = (taskIdx * rbs * nCoeff);
		for(int i = 0; i < rbs; i++){
			DA[dpOffset + i] = 1.0f; // power 0
		}
		for(int i = 0; i < rbs; i++){
			DA[dpOffset + i + rbs] = 0.0f; // power 1
		}
		
		int dStartIdx = taskIdx % (nBatch/2);
		for(int ds = 1; ds <= nDScale; ds++){
			// vec sumation
			int mapDStart = dStartIdx + (nDScale - ds) * (rbs/2);
			int dScale = dBaseScale * ds; // base_scale * current_scale
			for(int i = 0; i < dScale; i++){
				for(int j = 0; j < rbs; j++){
					if(taskIdx < (nBatch/2)){
						DA[dpOffset + rbs*ds + j] = DA[dpOffset + rbs*ds + j] + data[mapDStart + j*dScale + i];
					}else{ // gen reverse domain
						DA[dpOffset + rbs*ds + j] = DA[dpOffset + rbs*ds + j] + dataRev[mapDStart + j*dScale+ i];
					}
				}
			}
	
			// vec scalig
			for(int j = 0; j < rbs; j++){
				DA[dpOffset + rbs*ds + j] = DA[dpOffset + rbs*ds + j]/dScale;
			}
		}
		
		// calculate next degree
		for(int j = 2; j < nDegree; j++){
			int degreePad = (j * rbs * nDScale );
			for(int i = 0; i < rbs * nDScale; i++){
				DA[i + dpOffset + rbs + degreePad] = DA[j + dpOffset + rbs] * DA[j + dpOffset + rbs + degreePad - rbs] ; // power n>=2
			}
		}

		// initialize covariance matrix with regularization
		int apOffset = (taskIdx * nCoeff * nCoeff);
		for(int i = 0; i < nCoeff * nCoeff; i+= nCoeff+1){
			AA[apOffset + i] = regularize * regularize; // power 0
		}

		// pointing section
		DP[taskIdx] = (DA + taskIdx * rbs * nCoeff);
		AP[taskIdx] = (AA + taskIdx * nCoeff * nCoeff);
		IP[taskIdx] = (IA + taskIdx * nCoeff * nCoeff);
    }
}

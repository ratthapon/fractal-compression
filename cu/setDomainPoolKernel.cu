
#include <hip/hip_runtime.h>
extern "C"
__global__ void setDomainPoolKernel(
  int nBatch,int rbs,int nDegree,int nD,int dScale, float regularize,

  float *data,float *dataRev, // array of data and reverse data
  // arrays pointer
  float *DA,
  float *AA,
  float *IA,
  // pointer of array of pointer to pointer of array in arrays, nevermind i just stun you.
  // p(i) = data(i + size(data))
  float **DP,
  float **AP,
  float **IP
)
{
  int taskIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (taskIdx < nBatch){
    // initialize domain arrays
    // array structure
    // DA = rbs-rows , 1 + D_1^1 + D_2^1 + D_3^1 + ... + D_ds^2
    // nCoeff is power of Domains. start from power 0
    // nDegree determine the number of degree (maximum degree)
    // nD determine the number of domain blocks
    // dScale determing the scale of domain size compare to rbs
    // dpOffset determine the number of bytes padded for each Array

    int nCoeff = ((nDegree - 1) * nD + 1);

    // pointing array of pointers to array
    const int daOffset = taskIdx * rbs * nCoeff;
    const int aaOffset = taskIdx * nCoeff * nCoeff;
    const int iaOffset = taskIdx * nCoeff * nCoeff;

    DP[taskIdx] = &DA[daOffset];
    AP[taskIdx] = &AA[aaOffset];
    IP[taskIdx] = &IA[iaOffset];

    // initialize covariance matrix with regularization
    for(int i = 0; i < nCoeff * nCoeff; i++){
      AA[aaOffset + i] = 0.0f;
    }
    for(int i = 0; i < nCoeff * nCoeff; i+= nCoeff+1){
      // set diagonal to regularization parameter
      AA[aaOffset + i] = regularize * regularize;
    }

    // initialize first column covariance matrix
    for(int i = 0; i < rbs; i++){
      DA[daOffset + i] = 1.0f; // power 0
    }

    int dIdx = taskIdx % (nBatch/2);

    // for each block number dn
    for(int dn = 1; dn <= nD; dn++){
      // set reference domain block
      int dSize = rbs * dScale;
      int dnIdx = dIdx + (dn - 1) * dSize; // * domian location factor

      int rowDA = rbs*dn; // number of row of DA

      // initialize column dn-th index
      for(int i = 0; i < rbs; i++){
        DA[daOffset + rowDA + i] = 0.0f; // power 1
      }

      // construct DA from domain blocks at power 1
      // copy elements
      for(int i = 0; i < rbs; i++){
        if(taskIdx < (nBatch/2)){
          DA[daOffset + rowDA + i] =
          DA[daOffset + rowDA + i] + data[dnIdx + i*dScale];
        }else{ // gen reverse domain
          DA[daOffset + rowDA + i] =
          DA[daOffset + rowDA + i] + dataRev[dnIdx + i*dScale];
        }
      }

      // handling if domain blocks are larger than rbs (by downsample)
      for(int ds = 1; ds < dScale; ds++){
        // vec sumation
        for(int i = 0; i < rbs; i++){
          if(taskIdx < (nBatch/2)){
            DA[daOffset + rowDA + i] =
            DA[daOffset  + rowDA + i] + data[dnIdx + ds + i*dScale];
          }else{ // gen reverse domain
            DA[daOffset + rowDA + i] =
            DA[daOffset  + rowDA + i] + dataRev[dnIdx + ds + i*dScale];
          }
        }
      }

      // vec scalig after resample
      for(int i = 0; i < rbs; i++){
        DA[daOffset + rowDA + i] = DA[daOffset + rowDA + i]/dScale;
      }

      // calculate next degree
      // for(int deg = 2; deg <= nDegree; deg++){
      //   int degPad = rowDA * nD * (deg - 2);
      //   int nextDegPad = rowDA * nD * (deg - 1);
      //   for(int i = 0; i < rbs * nD; i++){
      //     // power n>=2
      //     // D^n = D^1 * D^(n-1)
      //     DA[daOffset + nextDegPad + rbs + i] =
      //     DA[daOffset + rbs + i] * DA[daOffset + degPad + rbs + i] ;
      //   }
      // }
    }
  }
}

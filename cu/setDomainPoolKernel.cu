
#include <hip/hip_runtime.h>
extern "C"
__device__ int getDnIdx(int dIdx, int dn, int nD, int rbs, int dScale, int expansion, bool isCenAlign);

__global__ void setDomainPoolKernel(
  int nBatch,int rbs,int nDegree,int nD,int dScale,int expansion,bool isCenAlign, float regularize,

  float *data,float *dataRev, // array of data and reverse data
  // arrays pointer
  float *DA,
  float *AA,
  float *IA,
  // pointer of array of pointer to pointer of array in arrays, nevermind i just stun you.
  // p(i) = data(i + size(data))
  float **DP,
  float **AP,
  float **IP
)
{
  int taskIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (taskIdx < nBatch){
    // initialize domain arrays
    // array structure
    // DA = rbs-rows , 1 + D_1^1 + D_2^1 + D_3^1 + ... + D_ds^2
    // nCoeff is power of Domains. start from power 0
    // nDegree determine the number of degree (maximum degree + 1)
    // nD determine the number of domain blocks
    // dScale determing the scale of domain size compare to rbs
    // dpOffset determine the number of bytes padded for each Array

    int nCoeff = ((nDegree - 1) * nD + 1);

    // pointing array of pointers to array
    const int daOffset = taskIdx * rbs * nCoeff;
    const int aaOffset = taskIdx * nCoeff * nCoeff;
    const int iaOffset = taskIdx * nCoeff * nCoeff;

    DP[taskIdx] = &DA[daOffset];
    AP[taskIdx] = &AA[aaOffset];
    IP[taskIdx] = &IA[iaOffset];

    // initialize covariance matrix with regularization
    for(int i = 0; i < nCoeff * nCoeff; i++){
      AA[aaOffset + i] = 0.0f;
    }
    for(int i = 0; i < nCoeff * nCoeff; i+= nCoeff+1){
      // set diagonal to regularization parameter
      AA[aaOffset + i] = regularize * regularize;
    }

    // initialize first column covariance matrix
    for(int i = 0; i < rbs; i++){
      DA[daOffset + i] = 1.0f; // power 0
    }

    int dIdx = taskIdx % (nBatch/2);

    // for each block number dn
    for(int dn = 1; dn <= nD; dn++){
      // set reference domain block

      int dnIdx = getDnIdx(dIdx, dn, nD, rbs, dScale, expansion, isCenAlign);
      //int dnIdx = dIdx + rbs * sumScale; // * domian location factor
      int dnScale = (int) powf( (float) dScale, (float) (1 + expansion * (dn - 1)));

      int padDA = rbs*dn; // number of row of DA

      // initialize column dn-th index
      for(int i = 0; i < rbs; i++){
        DA[daOffset + padDA + i] = 0.0f; // power 1
      }

      // construct DA from domain blocks at power 1
      // copy elements
      for(int i = 0; i < rbs; i++){
        if(taskIdx < (nBatch/2)){
          DA[daOffset + padDA + i] =
          DA[daOffset + padDA + i] + data[dnIdx + i*dnScale];
        }else{ // gen reverse domain
          DA[daOffset + padDA + i] =
          DA[daOffset + padDA + i] + dataRev[dnIdx + i*dnScale];
        }
      }

      // handling if domain blocks are larger than rbs (by downsample)
      for(int ds = 1; ds < dnScale; ds++){
        // vec sumation
        for(int i = 0; i < rbs; i++){
          if(taskIdx < (nBatch/2)){
            DA[daOffset + padDA + i] =
            DA[daOffset  + padDA + i] + data[dnIdx + ds + i*dnScale];
          }else{ // gen reverse domain
            DA[daOffset + padDA + i] =
            DA[daOffset  + padDA + i] + dataRev[dnIdx + ds + i*dnScale];
          }
        }
      }

      // vec scalig after resample
      for(int i = 0; i < rbs; i++){
        DA[daOffset + padDA + i] = DA[daOffset + padDA + i]/dnScale;
      }

      // calculate next degree
      for(int deg = 2; deg <= nDegree - 1; deg++){
        int degPad = rbs * nD * (deg - 2) + rbs * dn;
        int nextDegPad = rbs * nD * (deg - 1) + rbs * dn;
        for(int i = 0; i < rbs; i++){
          // power n>=2
          // D^n = D^1 * D^(n-1)
          DA[daOffset + nextDegPad + i] =
          DA[daOffset + rbs*dn + i] * DA[daOffset + degPad + i] ;
        }
      }
    }
  }
}

__device__ int getDnIdx(int dIdx, int dn, int nD, int rbs, int dScale, int expansion, bool isCenAlign){
  // compute sumScale
  int sumScale = 0;
  for(int k = 1; k <= nD && k < dn; k++){
    sumScale += (int) powf( (float) dScale, (float) (1 + expansion * (k - 1))) ;
  }
  int dnIdx = dIdx;
  if( !isCenAlign ){
    dnIdx = dIdx + rbs * sumScale;
  }
  return dnIdx;
}
